#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "DeviceFilter.cuh"

#include <stdio.h>

__global__ void DeviceGrayFilter(
	unsigned char* inputData,
	unsigned char* outputData,
	const int w,
	const int h,
	const int zoomedWidth,
	const int zoomedHeight,
	const int inPitch,
	const int outPitch
)
{
	const int x = blockIdx.x * THREADS_X + threadIdx.x;
	const int y = blockIdx.y * THREADS_Y + threadIdx.y;

	const int int_widht = inPitch / sizeof(int);
	const int width_border = (w + sizeof(int) - 1) / sizeof(int);

	uchar4* thread_input = (uchar4*)(inputData);
	uchar4* thread_output = (uchar4*)(outputData);

	__shared__ uchar4 shared_memory[THREADS_Y + 2][THREADS_X + 2];
	__shared__ uchar4 left_extension[1][THREADS_Y + 2];
	__shared__ uchar4 right_extension[1][THREADS_Y + 2];
	__shared__ uchar4 top_extension[1][THREADS_X + 2];
	__shared__ uchar4 bottom_extension[1][THREADS_X + 2];
	__shared__ uchar4 corners[4];

	if (y <= h)
	{
		uchar4* thread_input = (uchar4*)(inputData);
		uchar4* thread_output = (uchar4*)(outputData);

		shared_memory[threadIdx.y][threadIdx.x] = thread_input[y * int_widht + x];
		if (threadIdx.y == 0 && (blockIdx.y > 0))
		{
			top_extension[0][threadIdx.x] = thread_input[(y - 1) * int_widht + x];
		}

		if (threadIdx.y == THREADS_Y - 1)
		{
			bottom_extension[0][threadIdx.x] = thread_input[(y + 1) * int_widht + x];
		}

		if (y == h)
		{
			bottom_extension[0][threadIdx.x] = thread_input[(y)*int_widht + x];
		}

		if (threadIdx.x == 0 && blockIdx.x > 0)
		{
			left_extension[0][threadIdx.y] = thread_input[y * int_widht + x - 1];
		}

		if (threadIdx.x == THREADS_X - 1)
		{
			right_extension[0][threadIdx.y] = thread_input[y * int_widht + x + 1];
		}

		if (x != 0 && y != 0 && threadIdx.x == 0 && threadIdx.y == 0)
		{
			corners[0] = thread_input[(y - 1) * int_widht + x - 1];
		}

		if (y != 0 && threadIdx.y == 0 && threadIdx.x == THREADS_X - 1)
		{
			corners[1] = thread_input[(y - 1) * int_widht + x + 1];
		}

		if (x != 0 && threadIdx.y == THREADS_Y - 1 && threadIdx.x == 0)
		{
			corners[2] = thread_input[(y + 1) * int_widht + x - 1];
		}


		if (threadIdx.y == THREADS_Y - 1 && threadIdx.x == THREADS_X - 1)
		{
			corners[3] = thread_input[(y + 1) * int_widht + x + 1];
		}

		if (y == h && threadIdx.x == THREADS_X - 1)
		{
			corners[3] = thread_input[(y)*int_widht + x + 1];
		}


		if (threadIdx.y < 2)
		{
			if (y + THREADS_Y <= h)
			{
				shared_memory[THREADS_Y + threadIdx.y][threadIdx.x] = thread_input[(THREADS_Y + y) * int_widht + x];
			}
			if (threadIdx.x < THREADS_Y + 2)
			{
				int temp_x = blockIdx.x * THREADS_X + threadIdx.y;
				int temp_y = blockIdx.y * THREADS_Y + threadIdx.x;

				if (temp_x < int_widht && temp_y < zoomedHeight)
				{
					shared_memory[threadIdx.x][THREADS_X + threadIdx.y] = thread_input[temp_y * int_widht + THREADS_X + temp_x];
				}
			}
		}
	}

	__syncthreads();

	if (x <= int_widht && y <= h && y >= 1)
	{
		uchar4 generated_int = { 0 };
		unsigned char result;
		unsigned char value;
		uchar4 int_1;
		uchar4 mem;
		if (threadIdx.x == 0 && threadIdx.y == 0)
		{
			int_1.w = top_extension[0][0].x;
		}
		else if (threadIdx.x == 0)
		{
			int_1.w = shared_memory[threadIdx.y - 1][threadIdx.x].x;
		}
		else if (threadIdx.y == 0)
		{
			int_1.w = top_extension[0][threadIdx.x].x;
		}
		else
		{
			int_1.w = shared_memory[threadIdx.y - 1][threadIdx.x].x;
		}
		result = 0;
		value = int_1.w;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_1.w = result;
		uchar4 int_2;
		if (threadIdx.y == 0)
		{
			mem = top_extension[0][threadIdx.x];
			int_2.x = mem.y;
			int_2.y = mem.z;
			int_2.z = mem.w;
			int_2.w = threadIdx.x == THREADS_X - 1 ? corners[1].x : top_extension[0][threadIdx.x + 1].x;
		}
		else
		{
			mem = shared_memory[threadIdx.y - 1][threadIdx.x];
			int_2.x = mem.y;
			int_2.y = mem.z;
			int_2.z = mem.w;
			int_2.w = shared_memory[threadIdx.y - 1][threadIdx.x + 1].x;
		}
		result = 0;
		value = int_2.x;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_2.x = result;
		result = 0;
		value = int_2.y;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_2.y = result;
		result = 0;
		value = int_2.z;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_2.z = result;
		result = 0;
		value = int_2.w;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_2.w = result;
		uchar4 int_3;
		if (threadIdx.x == 0)
		{
			int_3.w = shared_memory[threadIdx.y][threadIdx.x].x;
		}
		else
		{
			int_3.w = shared_memory[threadIdx.y][threadIdx.x].x;
		}
		result = 0;
		value = int_3.w;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_3.w = result;
		uchar4 int_4;
		mem = shared_memory[threadIdx.y][threadIdx.x];
		int_4.x = mem.y;
		int_4.y = mem.z;
		int_4.z = mem.w;
		int_4.w = threadIdx.x == THREADS_X - 1 ? right_extension[0][threadIdx.y].x : shared_memory[threadIdx.y][threadIdx.x + 1].x;
		result = 0;
		value = int_4.x;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_4.x = result;
		result = 0;
		value = int_4.y;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_4.y = result;
		result = 0;
		value = int_4.z;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_4.z = result;
		result = 0;
		value = int_4.w;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_4.w = result;
		uchar4 int_5;
		int_5.w = threadIdx.y == THREADS_Y - 1 || y == h ? bottom_extension[0][threadIdx.x].x : shared_memory[threadIdx.y + 1][threadIdx.x].x;

		result = 0;
		value = int_5.w;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_5.w = result;
		uchar4 int_6;
		mem = threadIdx.y == THREADS_Y - 1 || y == h ? bottom_extension[0][threadIdx.x] : shared_memory[threadIdx.y + 1][threadIdx.x];
		int_6.x = mem.y;
		int_6.y = mem.z;
		int_6.z = mem.w;
		if (threadIdx.x == THREADS_X - 1)
		{
			int_6.w = threadIdx.y == THREADS_Y - 1 || y == h ? corners[3].x : shared_memory[threadIdx.y + 1][threadIdx.x + 1].x;
		}
		else
		{
			int_6.w = threadIdx.y == THREADS_Y - 1 || y == h ? bottom_extension[0][threadIdx.x + 1].x : shared_memory[threadIdx.y + 1][threadIdx.x + 1].x;
		}

		result = 0;
		value = int_6.x;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_6.x = result;
		result = 0;
		value = int_6.y;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_6.y = result;
		result = 0;
		value = int_6.z;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_6.z = result;
		result = 0;
		value = int_6.w;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_6.w = result;
		uchar4 int_7;
		if (threadIdx.y == 0)
		{
			mem = threadIdx.x >= THREADS_X - 1 ? corners[1] : top_extension[0][threadIdx.x + 1];
			int_7.x = mem.y;
		}
		else
		{
			mem = shared_memory[threadIdx.y - 1][threadIdx.x + 1];
			int_7.x = mem.y;
		}
		result = 0;
		value = int_7.x;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_7.x = result;
		uchar4 int_8;
		mem = shared_memory[threadIdx.y][threadIdx.x + 1];
		int_8.x = mem.y;
		result = 0;
		value = int_8.x;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_8.x = result;
		uchar4 int_9;
		if (threadIdx.y == THREADS_Y - 1 || y == h)
		{
			mem = threadIdx.x == THREADS_X - 1 ? corners[3] : bottom_extension[0][threadIdx.x + 1];
		}
		else
		{
			mem = shared_memory[threadIdx.y + 1][threadIdx.x + 1];
		}

		int_9.x = mem.y;
		result = 0;
		value = int_9.x;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_9.x = result;

		generated_int.x = (
			int_1.w + int_2.x * (-2) + int_2.y +
			int_3.w * (-2) + int_4.x * 4 + int_4.y * (-2) +
			int_5.w + int_6.x * (-2) + int_6.y
			);

		generated_int.y = (
			int_2.x + int_2.y * (-2) + int_2.z +
			int_4.x * (-2) + int_4.y * 4 + int_4.z * (-2) +
			int_6.x + int_6.y * (-2) + int_6.z
			);

		generated_int.z = (
			int_2.y + int_2.z * (-2) + int_2.w +
			int_4.y * (-2) + int_4.z * 4 + int_4.w * (-2) +
			int_6.y + int_6.z * (-2) + int_6.w
			);

		generated_int.w = (
			int_2.z + int_2.w * (-2) + int_7.x +
			int_4.z * (-2) + int_4.w * 4 + int_8.x * (-2) +
			int_6.z + int_6.w * (-2) + int_9.x
			);

		const int output_int_width = outPitch / sizeof(int);


		result = 0;
		value = generated_int.x;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		generated_int.x = result;
		result = 0;
		value = generated_int.y;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		generated_int.y = result;
		result = 0;
		value = generated_int.z;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		generated_int.z = result;
		result = 0;
		value = generated_int.w;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		generated_int.w = result;
		thread_output[(y - 1) * output_int_width + x] = generated_int;
	}
}

__global__ void DeviceRgbFilter(
	unsigned char* inputData,
	unsigned char* outputData,
	const int w,
	const int h,
	const int zoomedWidth,
	const int zoomedHeight,
	const int inPitch,
	const int outPitch
)
{
	const int x = blockIdx.x * THREADS_X + threadIdx.x;
	const int y = blockIdx.y * THREADS_Y + threadIdx.y;

	const int int_widht = inPitch / sizeof(int);
	const int width_border = (w + sizeof(int) - 1) / sizeof(int);

	uchar4* thread_input = (uchar4*)(inputData);
	uchar4* thread_output = (uchar4*)(outputData);

	__shared__ uchar4 shared_memory[THREADS_Y + 2][THREADS_X + 2];
	__shared__ uchar4 left_extension[1][THREADS_Y + 2];
	__shared__ uchar4 right_extension[1][THREADS_Y + 2];
	__shared__ uchar4 top_extension[1][THREADS_X + 2];
	__shared__ uchar4 bottom_extension[1][THREADS_X + 2];
	__shared__ uchar4 corners[4];

	if (y <= h)
	{
		uchar4* thread_input = (uchar4*)(inputData);
		uchar4* thread_output = (uchar4*)(outputData);

		shared_memory[threadIdx.y][threadIdx.x] = thread_input[y * int_widht + x];
		if (threadIdx.y == 0 && (blockIdx.y > 0))
		{
			top_extension[0][threadIdx.x] = thread_input[(y - 1) * int_widht + x];
		}

		if (threadIdx.y == THREADS_Y - 1)
		{
			bottom_extension[0][threadIdx.x] = thread_input[(y + 1) * int_widht + x];
		}

		if (y == h)
		{
			bottom_extension[0][threadIdx.x] = thread_input[(y)*int_widht + x];
		}

		if (threadIdx.x == 0 && blockIdx.x > 0)
		{
			left_extension[0][threadIdx.y] = thread_input[y * int_widht + x - 1];
		}

		if (threadIdx.x == THREADS_X - 1)
		{
			right_extension[0][threadIdx.y] = thread_input[y * int_widht + x + 1];
		}

		if (x != 0 && y != 0 && threadIdx.x == 0 && threadIdx.y == 0)
		{
			corners[0] = thread_input[(y - 1) * int_widht + x - 1];
		}

		if (y != 0 && threadIdx.y == 0 && threadIdx.x == THREADS_X - 1)
		{
			corners[1] = thread_input[(y - 1) * int_widht + x + 1];
		}

		if (x != 0 && threadIdx.y == THREADS_Y - 1 && threadIdx.x == 0)
		{
			corners[2] = thread_input[(y + 1) * int_widht + x - 1];
		}


		if (threadIdx.y == THREADS_Y - 1 && threadIdx.x == THREADS_X - 1)
		{
			corners[3] = thread_input[(y + 1) * int_widht + x + 1];
		}

		if (y == h && threadIdx.x == THREADS_X - 1)
		{
			corners[3] = thread_input[(y)*int_widht + x + 1];
		}


		if (threadIdx.y < 2)
		{
			if (y + THREADS_Y <= h)
			{
				shared_memory[THREADS_Y + threadIdx.y][threadIdx.x] = thread_input[(THREADS_Y + y) * int_widht + x];
			}
			if (threadIdx.x < THREADS_Y + 2)
			{
				int temp_x = blockIdx.x * THREADS_X + threadIdx.y;
				int temp_y = blockIdx.y * THREADS_Y + threadIdx.x;

				if (temp_x < int_widht && temp_y < zoomedHeight)
				{
					shared_memory[threadIdx.x][THREADS_X + threadIdx.y] = thread_input[temp_y * int_widht + THREADS_X + temp_x];
				}
			}
		}
	}

	__syncthreads();

	if (x <= int_widht && y <= h && y >= 1)
	{
		uchar4 generated_int = { 0 };
		unsigned char result;
		unsigned char value;
		uchar4 int_1;
		uchar4 mem;
		if (threadIdx.x == 0 && threadIdx.y == 0)
		{
			mem = corners[0];
			int_1.x = mem.w;
			mem = top_extension[0][0];
			int_1.y = mem.x;
			int_1.z = mem.y;
			int_1.w = mem.z;
		}
		else if (threadIdx.x == 0)
		{
			mem = left_extension[0][threadIdx.y];
			int_1.x = mem.w;
			mem = shared_memory[threadIdx.y - 1][threadIdx.x];
			int_1.y = mem.x;
			int_1.z = mem.y;
			int_1.w = mem.z;
		}
		else if (threadIdx.y == 0)
		{
			mem = top_extension[0][threadIdx.x - 1];
			int_1.x = mem.w;
			mem = top_extension[0][threadIdx.x];
			int_1.y = mem.x;
			int_1.z = mem.y;
			int_1.w = mem.z;
		}
		else
		{
			mem = shared_memory[threadIdx.y - 1][threadIdx.x - 1];
			int_1.x = mem.w;
			mem = shared_memory[threadIdx.y - 1][threadIdx.x];
			int_1.y = mem.x;
			int_1.z = mem.y;
			int_1.w = mem.z;
		}
		result = 0;
		value = int_1.x;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_1.x = result;
		result = 0;
		value = int_1.y;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_1.y = result;
		result = 0;
		value = int_1.z;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_1.z = result;
		result = 0;
		value = int_1.w;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_1.w = result;
		uchar4 int_2;
		if (threadIdx.y == 0)
		{
			mem = top_extension[0][threadIdx.x];
			int_2.x = mem.w;
			mem = threadIdx.x == THREADS_X - 1 ? corners[1] : top_extension[0][threadIdx.x + 1];
			int_2.y = mem.x;
			int_2.z = mem.y;
			int_2.w = mem.z;
		}
		else
		{
			mem = shared_memory[threadIdx.y - 1][threadIdx.x];
			int_2.x = mem.w;
			mem = shared_memory[threadIdx.y - 1][threadIdx.x + 1];
			int_2.y = mem.x;
			int_2.z = mem.y;
			int_2.w = mem.z;
		}
		result = 0;
		value = int_2.x;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_2.x = result;
		result = 0;
		value = int_2.y;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_2.y = result;
		result = 0;
		value = int_2.z;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_2.z = result;
		result = 0;
		value = int_2.w;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_2.w = result;
		uchar4 int_3;
		if (threadIdx.x == 0)
		{
			mem = left_extension[0][threadIdx.y];
			int_3.x = mem.w;
			mem = shared_memory[threadIdx.y][threadIdx.x];
			int_3.y = mem.x;
			int_3.z = mem.y;
			int_3.w = mem.z;
		}
		else
		{
			mem = shared_memory[threadIdx.y][threadIdx.x - 1];
			int_3.x = mem.w;
			mem = shared_memory[threadIdx.y][threadIdx.x];
			int_3.y = mem.x;
			int_3.z = mem.y;
			int_3.w = mem.z;
		}
		result = 0;
		value = int_3.x;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_3.x = result;
		result = 0;
		value = int_3.y;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_3.y = result;
		result = 0;
		value = int_3.z;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_3.z = result;
		result = 0;
		value = int_3.w;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_3.w = result;
		uchar4 int_4;
		mem = shared_memory[threadIdx.y][threadIdx.x];
		int_4.x = mem.w;
		mem = threadIdx.x == THREADS_X - 1 ? right_extension[0][threadIdx.y] : shared_memory[threadIdx.y][threadIdx.x + 1];
		int_4.y = mem.x;
		int_4.z = mem.y;
		int_4.w = mem.z;
		result = 0;
		value = int_4.x;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_4.x = result;
		result = 0;
		value = int_4.y;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_4.y = result;
		result = 0;
		value = int_4.z;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_4.z = result;
		result = 0;
		value = int_4.w;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_4.w = result;
		uchar4 int_5;
		mem = shared_memory[threadIdx.y][threadIdx.x];
		int_5.x = mem.w;
		mem = threadIdx.y == THREADS_Y - 1 || y == h ? bottom_extension[0][threadIdx.x] : shared_memory[threadIdx.y + 1][threadIdx.x];
		int_5.y = mem.x;
		int_5.z = mem.y;
		int_5.w = mem.z;
		result = 0;
		value = int_5.x;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_5.x = result;
		result = 0;
		value = int_5.y;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_5.y = result;
		result = 0;
		value = int_5.z;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_5.z = result;
		result = 0;
		value = int_5.w;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_5.w = result;
		uchar4 int_6;
		mem = threadIdx.y == THREADS_Y - 1 || y == h ? bottom_extension[0][threadIdx.x] : shared_memory[threadIdx.y + 1][threadIdx.x];
		int_6.x = mem.w;
		
		if (threadIdx.x == THREADS_X - 1)
		{
			mem = threadIdx.y == THREADS_Y - 1 || y == h ? corners[3] : shared_memory[threadIdx.y + 1][threadIdx.x + 1];
		}
		else
		{
			mem = threadIdx.y == THREADS_Y - 1 || y == h ? bottom_extension[0][threadIdx.x + 1] : shared_memory[threadIdx.y + 1][threadIdx.x + 1];
		}
		int_6.y = mem.x;
		int_6.z = mem.y;
		int_6.w = mem.z;
		result = 0;
		value = int_6.x;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_6.x = result;
		result = 0;
		value = int_6.y;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_6.y = result;
		result = 0;
		value = int_6.z;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_6.z = result;
		result = 0;
		value = int_6.w;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_6.w = result;
		uchar4 int_7;
		if (threadIdx.y == 0)
		{
			mem = threadIdx.x >= THREADS_X - 1 ? corners[1] : top_extension[0][threadIdx.x + 1];
			int_7.x = mem.w;
			mem = top_extension[0][threadIdx.x + 2];
			int_7.y = mem.x;
			int_7.z = mem.y;
			int_7.w = mem.z;
		}
		else
		{
			mem = shared_memory[threadIdx.y - 1][threadIdx.x + 1];
			int_7.x = mem.w;
			mem = shared_memory[threadIdx.y - 1][threadIdx.x + 2];
			int_7.y = mem.x;
			int_7.z = mem.y;
			int_7.w = mem.z;
		}
		result = 0;
		value = int_7.x;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_7.x = result;
		result = 0;
		value = int_7.y;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_7.y = result;
		result = 0;
		value = int_7.z;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_7.z = result;
		result = 0;
		value = int_7.w;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_7.w = result;
		uchar4 int_8;
		mem = shared_memory[threadIdx.y][threadIdx.x + 1];
		int_8.x = mem.w;
		mem = shared_memory[threadIdx.y][threadIdx.x + 2];
		int_8.y = mem.x;
		int_8.z = mem.y;
		int_8.w = mem.z;
		result = 0;
		value = int_8.x;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_8.x = result;
		result = 0;
		value = int_8.y;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_8.y = result;
		result = 0;
		value = int_8.z;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_8.z = result;
		result = 0;
		value = int_8.w;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_8.w = result;
		uchar4 int_9;
		if (threadIdx.y == THREADS_Y - 1 || y == h)
		{
			mem = threadIdx.x == THREADS_X - 1 ? corners[3] : bottom_extension[0][threadIdx.x + 1];
		}
		else
		{
			mem = shared_memory[threadIdx.y + 1][threadIdx.x + 1];
		}

		int_9.x = mem.w;
		mem = shared_memory[threadIdx.y + 1][threadIdx.x + 2];
		int_9.y = mem.x;
		int_9.z = mem.y;
		int_9.w = mem.z;
		result = 0;
		value = int_9.x;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_9.x = result;
		result = 0;
		value = int_9.y;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_9.y = result;
		result = 0;
		value = int_9.z;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_9.z = result;
		result = 0;
		value = int_9.w;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		int_9.w = result;

		generated_int.x = (
			int_1.y + int_2.x * (-2) + int_2.w +
			int_3.y * (-2) + int_4.x * 4 + int_4.w * (-2) +
			int_5.y + int_6.x * (-2) + int_6.w
			);

		generated_int.y = (
			int_1.z + int_2.y * (-2) + int_7.x +
			int_3.z * (-2) + int_4.y * 4 + int_8.x * (-2) +
			int_5.z + int_6.y * (-2) + int_9.x
			);

		generated_int.z = (
			int_1.w + int_2.z * (-2) + int_7.y +
			int_3.w * (-2) + int_4.z * 4 + int_8.y * (-2) +
			int_5.w + int_6.z * (-2) + int_9.y
			);

		generated_int.w = (
			int_2.x + int_2.w * (-2) + int_7.z +
			int_4.x * (-2) + int_4.w * 4 + int_8.z * (-2) +
			int_6.x + int_6.w * (-2) + int_9.z
			);

		const int output_int_width = outPitch / sizeof(int);


		result = 0;
		value = generated_int.x;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		generated_int.x = result;
		result = 0;
		value = generated_int.y;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		generated_int.y = result;
		result = 0;
		value = generated_int.z;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		generated_int.z = result;
		result = 0;
		value = generated_int.w;
		for (int g = 0; g < 8; g++)
		{
			result <<= 1;
			result |= (value & 1);
			value >>= 1;
		}
		generated_int.w = result;
		thread_output[(y - 1) * output_int_width + x] = generated_int;
	}
}