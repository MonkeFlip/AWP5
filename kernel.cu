#include <cstdlib>
#include <cstdio>
#include <cassert>
#include <iostream>
#include <iomanip>
#include <chrono>

#include "hip/hip_runtime.h"
#include ""
#include <helper_image.h>

#include "HostFilter.h"
#include "DeviceFilter.cuh"

using namespace std;

void CompareImages(
	const unsigned char* imageA,
	unsigned char* imageB,
	int size,
	int* difference
)
{
	*difference = 0;
	int counter = 0;
	for (int i = 0; i < size; i++)
	{
		if (imageA[i] != imageB[i])
		{
			//cout << i << ": " << imageA[i] << " " << imageB[i] << endl;
			//cout << " != ";
			//return;
			imageB[i] = 255;
			if (counter++ < 10)
			{
				cout << i << " ";
			}
			(*difference)++;
		}
	}

	cout << endl;
	if ((*difference) == 0)
	{
		cout << "Images are equal." << endl;;
	}
	else
	{
		cout << "Images are different." << endl;
	}
}

void StartHostGray(
	unsigned char* inputDataRgb,
	unsigned char* hostResult,
	size_t w,
	size_t h
)
{
	auto start_cpu = chrono::steady_clock::now();
	HostGrayFilter(inputDataRgb, hostResult, w, h);
	auto end_cpu = chrono::steady_clock::now();
	auto cpu_time = end_cpu - start_cpu;
	cout << "Host time (Gray): " << chrono::duration<double, milli>(cpu_time).count() << endl << endl;
}

void StartDeviceGray(
	unsigned char* inputDataGray,
	unsigned char* deviceResultGray,
	size_t w,
	size_t h,
	const size_t zoomedHeight,
	const size_t zoomedWidth
)
{
	size_t input_pitch = 0;
	unsigned char* zoomed_input = ExtendBorders(inputDataGray, w, h);
	unsigned char* pitched_input_data = nullptr;
	hipMallocPitch((void**)(&pitched_input_data), &input_pitch, zoomedWidth, zoomedHeight);
	hipMemcpy2D(
		pitched_input_data,
		input_pitch,
		zoomed_input,
		zoomedWidth,
		zoomedWidth,
		zoomedHeight,
		hipMemcpyHostToDevice
	);

	size_t output_pitch = 0;
	unsigned char* pitched_output_data = nullptr;
	hipMallocPitch((void**)(&pitched_output_data), &output_pitch, w, h + 1);

	float time = 0;
	hipEvent_t startEvent, stopEvent;
	hipEventCreate(&startEvent);
	hipEventCreate(&stopEvent);

	int grid_x = (input_pitch + THREADS_X - 1) / THREADS_X;
	int grid_y = (h + THREADS_Y - 1) / THREADS_Y + 1;
	dim3 dimGrid(grid_x, grid_y, 1);
	dim3 dimBlock(THREADS_X, THREADS_Y, 1);

	cout << "Grid size: " << dimGrid.x << " x " << dimGrid.y << endl;
	cout << "Block size: " << dimBlock.x << " x " << dimBlock.y << endl << endl;

	hipEventRecord(startEvent, 0);
	DeviceGrayFilter << <dimGrid, dimBlock >> > (
		pitched_input_data, pitched_output_data,
		w, h,
		zoomedWidth, zoomedHeight,
		input_pitch, output_pitch
		);

	hipEventRecord(stopEvent, 0);
	hipEventSynchronize(stopEvent);
	hipEventElapsedTime(&time, startEvent, stopEvent);

	cout << "Device time (Gray): " << time << endl << endl;

	hipMemcpy2D(
		deviceResultGray,
		w,
		pitched_output_data,
		output_pitch,
		w,
		h,
		hipMemcpyDeviceToHost
	);

	hipEventDestroy(startEvent);
	hipEventDestroy(stopEvent);
	hipFree(pitched_input_data);
	hipFree(pitched_output_data);
}

void StartHostRGB(
	RGB* inputDataRgb,
	RGB* hostResult,
	size_t w,
	size_t h
)
{
	auto start_cpu = chrono::steady_clock::now();
	HostRgbFilter(inputDataRgb, hostResult, w, h);
	auto end_cpu = chrono::steady_clock::now();
	auto cpu_time = end_cpu - start_cpu;
	cout << "Host time (RGB): " << chrono::duration<double, milli>(cpu_time).count() << endl << endl;
}

void StartDeviceRGB(
	RGB* inputDataRgb,
	RGB* deviceResult,
	size_t w,
	size_t h,
	const size_t zoomedWidthInBytes,
	const size_t zoomedHeight,
	const size_t widthInBytes
)
{
	size_t input_pitch = 0;
	RGB* zoomed_input = ExtendBorders(inputDataRgb, w, h);
	unsigned char* pitched_input_data = nullptr;
	hipMallocPitch((void**)(&pitched_input_data), &input_pitch, zoomedWidthInBytes, zoomedHeight);
	hipMemcpy2D(
		pitched_input_data,
		input_pitch,
		(unsigned char**)(zoomed_input),
		zoomedWidthInBytes,
		zoomedWidthInBytes,
		zoomedHeight,
		hipMemcpyHostToDevice
	);

	size_t output_pitch = 0;
	unsigned char* pitched_output_data = nullptr;
	hipMallocPitch((void**)(&pitched_output_data), &output_pitch, widthInBytes, h);

	float time = 0;
	hipEvent_t startEvent, stopEvent;
	hipEventCreate(&startEvent);
	hipEventCreate(&stopEvent);

	int grid_x = (input_pitch + THREADS_X - 1) / THREADS_X;
	int grid_y = (zoomedHeight + THREADS_Y - 1) / THREADS_Y;

	dim3 dimGrid(grid_x, grid_y, 1);
	dim3 dimBlock(THREADS_X, THREADS_Y, 1);

	cout << "Grid size: " << dimGrid.x << " x " << dimGrid.y << endl;
	cout << "Block size: " << dimBlock.x << " x " << dimBlock.y << endl << endl;

	hipEventRecord(startEvent, 0);
	DeviceRgbFilter << <dimGrid, dimBlock >> > (
		pitched_input_data, pitched_output_data,
		widthInBytes, h,
		zoomedWidthInBytes, zoomedHeight,
		input_pitch, output_pitch
		);
	hipEventRecord(stopEvent, 0);
	hipEventSynchronize(stopEvent);
	hipEventElapsedTime(&time, startEvent, stopEvent);

	cout << "Device time (RGB): " << time << endl << endl;

	hipMemcpy2D(
		(unsigned char*)(deviceResult),
		widthInBytes,
		pitched_output_data,
		output_pitch,
		widthInBytes,
		h,
		hipMemcpyDeviceToHost
	);

	hipEventDestroy(startEvent);
	hipEventDestroy(stopEvent);
	hipFree(pitched_input_data);
	hipFree(pitched_output_data);
}

int main()
{
	size_t w = 0;
	size_t h = 0;
	int channels = 0;
	string fileName = "images/amogus";
	unsigned char* inputDataGray = nullptr;
	__loadPPM(
		(fileName + ".pgm").c_str(),
		&inputDataGray,
		(unsigned int*)(&w),
		(unsigned int*)(&h),
		(unsigned int*)(&channels)
	);

	cout << "----------------Gray----------------" << endl;
	cout << fileName << ".pgm" << " (" << w << " x " << h << ") Channels: " << channels << endl << endl;

	const size_t zoomedWidth = w + 2;
	const size_t zoomedHeight = h + 2;
	const size_t size = w * h;

	unsigned char* hostResultGray = new unsigned char[size];
	unsigned char* deviceResultGray = new unsigned char[size];

	StartHostGray(
		inputDataGray,
		hostResultGray,
		w,
		h
	);

	StartDeviceGray(
		inputDataGray,
		deviceResultGray,
		w,
		h,
		zoomedHeight,
		zoomedWidth
	);

	int difference = 0;
	CompareImages(hostResultGray, deviceResultGray, size, &difference);
	cout << "Difference between images: " << difference << endl;

	__savePPM((fileName + "_HostGray.pgm").c_str(), hostResultGray, w, h, channels);
	__savePPM((fileName + "_DeviceGray.pgm").c_str(), deviceResultGray, w, h, channels);



	/*cout << "-----------------RGB----------------" << endl;
	RGB* inputDataRgb = nullptr;
	__loadPPM(
		(fileName + ".ppm").c_str(),
		(unsigned char**)(&inputDataRgb),
		(unsigned int*)(&w),
		(unsigned int*)(&h),
		(unsigned int*)(&channels)
	);

	const size_t widthInBytes = w * sizeof(RGB);
	const size_t zoomedWidthInBytes = zoomedWidth * sizeof(RGB);
	RGB* hostResultRGB = new RGB[size];
	RGB* deviceResultRGB = new RGB[size];

	cout << fileName << ".ppm" << " (" << w << " x " << h << ") Channels: " << channels << endl << endl;

	StartHostRGB(
		inputDataRgb,
		hostResultRGB,
		w,
		h
	);

	StartDeviceRGB(
		inputDataRgb,
		deviceResultRGB,
		w,
		h,
		zoomedWidthInBytes,
		zoomedHeight,
		widthInBytes
	);

	CompareImages((unsigned char*)(hostResultRGB), (unsigned char*)(deviceResultRGB), size, &difference);
	cout << "Difference between images: " << difference << endl;

	__savePPM((fileName + "_HostRGB.ppm").c_str(), (unsigned char*)(hostResultRGB), w, h, channels);
	__savePPM((fileName + "_DeviceRGB.ppm").c_str(), (unsigned char*)(deviceResultRGB), w, h, channels);*/

	delete[] inputDataGray;
	delete[] hostResultGray;
	delete[] deviceResultGray;

	/*delete[] inputDataRgb;
	delete[] hostResultRGB;
	delete[] deviceResultRGB;*/
}